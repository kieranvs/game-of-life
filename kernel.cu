
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void kernel(uint8_t* buf_current, uint8_t* buf_next, int dim)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int j = blockDim.y * blockIdx.y + threadIdx.y + 1;

	uint8_t neighbours = 0;
	neighbours += buf_current[i - 1 + dim * (j    )];
	neighbours += buf_current[i - 1 + dim * (j - 1)];
	neighbours += buf_current[i - 1 + dim * (j + 1)];

	neighbours += buf_current[i     + dim * (j - 1)];
	neighbours += buf_current[i     + dim * (j + 1)];

	neighbours += buf_current[i + 1 + dim * (j    )];
	neighbours += buf_current[i + 1 + dim * (j - 1)];
	neighbours += buf_current[i + 1 + dim * (j + 1)];

	if (buf_current[i + dim * j])
		buf_next[i + dim * j] = (neighbours == 2 || neighbours == 3);
	else
		buf_next[i + dim * j] = (neighbours == 3);
}

void run_kernel(uint8_t* buf_current, uint8_t* buf_next, int dim)
{
	int grid_sz = (dim - 2) / 32;
	dim3 block_dim(32, 32, 1);
	dim3 grid_dim(grid_sz, grid_sz, 1);
	kernel<<<grid_dim, block_dim>>>(buf_current, buf_next, dim);
}